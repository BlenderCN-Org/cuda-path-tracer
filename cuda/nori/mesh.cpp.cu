#include "hip/hip_runtime.h"
/*
    This file is part of Nori, a simple educational ray tracer

    Copyright (c) 2015 by Wenzel Jakob, Romain Prévost

    Nori is free software; you can redistribute it and/or modify
    it under the terms of the GNU General Public License Version 3
    as published by the Free Software Foundation.

    Nori is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program. If not, see <http://www.gnu.org/licenses/>.
*/

#include <nori/mesh.h>
#include <nori/bbox.h>
#include <nori/bsdf.h>
#include <nori/emitter.h>
#include <nori/warp.h>
#include <nori/normalModifier.h>
#include <nori/bumpMap.h>
#include <nori/normalMap.h>
#include <Eigen/Geometry>
#include <hip/hip_runtime_api.h>

NORI_NAMESPACE_BEGIN

Mesh::Mesh() : m_V(MatrixXf().data(),0,0),m_N(NULL,0,0),m_UV(NULL,0,0),m_F(NULL,0,0) {
    shapeType = ETMesh;
}

void Mesh::activate() {
    Shape::activate();

    m_pdf.reserve(getPrimitiveCount());
    for(uint32_t i = 0 ; i < getPrimitiveCount() ; ++i) {
        m_pdf.append(surfaceArea(i));
    }
    m_pdf.normalize();
    /**
     * make the class usable on cpu
     */
    //this->m_V =  m_V;
    //this->m_N = m_N;
    //this->m_UV = m_UV;
    //this->m_F = m_F;

}

__device__ void Mesh::sampleSurface(ShapeQueryRecord & sRec, const Point2f & sample) const {
    Point2f s = sample;

    size_t idT = m_pdf.sampleReuse(s.x());

    Vector3f bc = Warp::squareToUniformTriangle(s);

    //printf("bc norm_ s==%.3f \n", s.norm());

    sRec.p = getInterpolatedVertex(idT,bc);
    if (m_N.size() > 0) {
        sRec.n = getInterpolatedNormal(idT, bc);
    }
    else {
        Point3f p0 = m_V.col(m_F(0, idT));
        Point3f p1 = m_V.col(m_F(1, idT));
        Point3f p2 = m_V.col(m_F(2, idT));
        Normal3f n = (p1-p0).cross(p2-p0).normalized();
        sRec.n = n;
    }
    sRec.pdf = m_pdf.getNormalization();
}
__device__ float Mesh::pdfSurface(const ShapeQueryRecord & sRec) const {
    return m_pdf.getNormalization();
}

__device__ Point3f Mesh::getInterpolatedVertex(uint32_t index, const Vector3f &bc) const {
    return (bc.x() * m_V.col(m_F(0, index)) +
            bc.y() * m_V.col(m_F(1, index)) +
            bc.z() * m_V.col(m_F(2, index)));
}

__device__ Normal3f Mesh::getInterpolatedNormal(uint32_t index, const Vector3f &bc) const {
    return (bc.x() * m_N.col(m_F(0, index)) +
            bc.y() * m_N.col(m_F(1, index)) +
            bc.z() * m_N.col(m_F(2, index))).normalized();
}

__device__ float Mesh::surfaceArea(uint32_t index) const {
    uint32_t i0 = m_F(0, index), i1 = m_F(1, index), i2 = m_F(2, index);

    const Point3f p0 = m_V.col(i0), p1 = m_V.col(i1), p2 = m_V.col(i2);

    return 0.5f * Vector3f((p1 - p0).cross(p2 - p0)).norm();
}

__device__ bool Mesh::rayIntersect(uint32_t index, const Ray3f &ray, float &u, float &v, float &t) const {

    uint32_t i0 = m_F(0, index), i1 = m_F(1, index), i2 = m_F(2, index);
    const Point3f p0 = m_V.col(i0), p1 = m_V.col(i1), p2 = m_V.col(i2);

    /* Find vectors for two edges sharing v[0] */
    Vector3f edge1 = p1 - p0, edge2 = p2 - p0;

    /* Begin calculating determinant - also used to calculate U parameter */
    Vector3f pvec = ray.d.cross(edge2);

    /* If determinant is near zero, ray lies in plane of triangle */
    float det = edge1.dot(pvec);

    if (det > -1e-8f && det < 1e-8f)
        return false;
    float inv_det = 1.0f / det;

    /* Calculate distance from v[0] to ray origin */
    Vector3f tvec = ray.o - p0;

    /* Calculate U parameter and test bounds */
    u = tvec.dot(pvec) * inv_det;
    if (u < 0.0 || u > 1.0)
        return false;

    /* Prepare to test V parameter */
    Vector3f qvec = tvec.cross(edge1);

    /* Calculate V parameter and test bounds */
    v = ray.d.dot(qvec) * inv_det;
    if (v < 0.0 || u + v > 1.0)
        return false;

    /* Ray intersects triangle -> compute t */
    t = edge2.dot(qvec) * inv_det;

    return t >= ray.mint && t <= ray.maxt;
}

__device__ void Mesh::setHitInformation(uint32_t index, const Ray3f &ray, Intersection & its) const {
    /* Find the barycentric coordinates */
    Vector3f bary;
    bary << 1-its.uv.sum(), its.uv;

    /* Vertex indices of the triangle */
    uint32_t idx0 = m_F(0, index), idx1 = m_F(1, index), idx2 = m_F(2, index);

    Point3f p0 = m_V.col(idx0), p1 = m_V.col(idx1), p2 = m_V.col(idx2);

    /* Compute the intersection positon accurately
       using barycentric coordinates */
    its.p = bary.x() * p0 + bary.y() * p1 + bary.z() * p2;

    /* Compute proper texture coordinates if provided by the mesh */
    if (m_UV.size() > 0)
        its.uv = bary.x() * m_UV.col(idx0) +
                 bary.y() * m_UV.col(idx1) +
                 bary.z() * m_UV.col(idx2);

    /* Compute the geometry frame */
    its.geoFrame = Frame((p1-p0).cross(p2-p0).normalized());

    if (m_N.size() > 0) {
        /* Compute the shading frame. Note that for simplicity,
           the current implementation doesn't attempt to provide
           tangents that are continuous across the surface. That
           means that this code will need to be modified to be able
           use anisotropic BRDFs, which need tangent continuity */

        its.shFrame = Frame(
                (bary.x() * m_N.col(idx0) +
                 bary.y() * m_N.col(idx1) +
                 bary.z() * m_N.col(idx2)).normalized());


    } else {
        its.shFrame = its.geoFrame;
    }


    if (m_normalModifier != NULL)
    {
        Frame f = its.shFrame;
        its.shFrame = Frame(CallNormalModifier(m_normalModifier, eval, its.uv, its.shFrame));
    }
}

 __host__ BoundingBox3f Mesh::getBoundingBox(uint32_t index) const {
    BoundingBox3f result(m_V.col(m_F(0, index)));
    result.expandBy(m_V.col(m_F(1, index)));
    result.expandBy(m_V.col(m_F(2, index)));
    return result;
}

 __host__ Point3f Mesh::getCentroid(uint32_t index) const {
    return (1.0f / 3.0f) *
        (m_V.col(m_F(0, index)) +
         m_V.col(m_F(1, index)) +
         m_V.col(m_F(2, index)));
}


std::string Mesh::toString() const {
    return tfm::format(
        "Mesh[\n"
        "  name = \"%s\",\n"
        "  vertexCount = %i,\n"
        "  triangleCount = %i,\n"
        "  bsdf = %s,\n"
        "  emitter = %s\n"
        "]",
        m_name,
        m_V.cols(),
        m_F.cols(),
        m_bsdf ? indent(m_bsdf->toString()) : std::string("null"),
        m_emitter ? indent(m_emitter->toString()) : std::string("null")
    );
}
void Mesh::gpuTransfer(NoriObject **objects) {
    Shape::gpuTransfer(objects);
    /*we assume that we do not wan't to accces most of the stuff again on the cpu so we copy it over to the gpu*/
    void *pV,*pN,*pUV,*pF;
    //step 1 allocate alle the storage
    hipMalloc(&pV,sizeof(float)*m_V.size());
    hipMalloc(&pN,sizeof(float)*m_N.size());
    hipMalloc(&pUV,sizeof(float)*m_UV.size());
    hipMalloc(&pF,sizeof(uint32_t)*m_F.size());
    //step 2 copy
    hipMemcpy(pV,m_V.data(),sizeof(float)*m_V.size(),hipMemcpyHostToDevice);
    hipMemcpy(pN,m_N.data(),sizeof(float)*m_N.size(),hipMemcpyHostToDevice);
    hipMemcpy(pUV,m_UV.data(),sizeof(float)*m_UV.size(),hipMemcpyHostToDevice);
    hipMemcpy(pF,m_F.data(),sizeof(uint32_t)*m_F.size(),hipMemcpyHostToDevice);
    //step 3 change data pointer
    /**
     * Object will be broken for cpu use afterwards as all pointer
     */
   new(&this->m_V) Eigen::Map<MatrixXf> ((float *) pV,m_V.rows(),m_V.cols());
   new(&this->m_N) Eigen::Map<MatrixXf> ((float *) pN,m_N.rows(),m_N.cols());
   new(&this->m_UV)Eigen::Map<MatrixXf> ((float *) pUV,m_UV.rows(),m_UV.cols());
   new(&this->m_F) Eigen::Map<MatrixXu> ((uint32_t *) pF,m_F.rows(),m_F.cols());

    //last part
    m_pdf.transferGpu();
};

NORI_NAMESPACE_END

